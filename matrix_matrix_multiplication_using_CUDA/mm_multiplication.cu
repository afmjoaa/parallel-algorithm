#include "hip/hip_runtime.h"
﻿#include <hip/hip_runtime.h>
#include <>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <assert.h>

#include <iostream>
#include <chrono>
#include <ctime>

__global__ void matrixMul(int* a, int* b, int* c, int n) {
	// Compute each thread's row
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	// Compute each thread's column
	int col = blockIdx.x * blockDim.x + threadIdx.x;

	int temp_sum = 0;
	// Boundary protection
	if ((row < n) && (col < n)) {
		// Iterate over row, and down column
		for (int k = 0; k < n; k++) {
			// Accumulate result for a single element
			temp_sum += a[row * n + k] * b[k * n + col];
		}
		// Assign result
		c[row * n + col] = temp_sum;
	}
}

// Initialization function for matrices
void matrix_init(int* a, int n) {
	for (int i = 0; i < n; i++) {
		for (int j = 0; j < n; j++) {
			a[i * n + j] = rand() % 100;
		}
	}
}

// This program multiplies two squire matrix. Therefore, a single constant for matrix dimension is sufficient
int dim;  // Default value 4096
// the block size for blocked matrix-matrix multiplication
int blockSize; // Default value 16

int main(int argc, const char* argv[]) {
	// read the matrix dimension length and block size from command line
	if (argc > 2) {
		dim = atoi(argv[1]);
		blockSize = atoi(argv[2]);
	}
	else {
		// Assigning default param values here.
		dim = 4096;
		blockSize = 16;
		printf("Assigning default value dim = %d and blockSize = %d", dim, blockSize);
	}

	// Taking the start time
	auto start = std::chrono::system_clock::now();

	// Matrix size of 4096 x 4096;
	int n = dim;

	// Size (in bytes) of matrix
	size_t bytes = n * n * sizeof(int);

	// Host pointers
	int* h_a, * h_b, * h_c;

	// Allocate host memory
	h_a = (int*)malloc(bytes);
	h_b = (int*)malloc(bytes);
	h_c = (int*)malloc(bytes);

	// Device pointers
	int* d_a, * d_b, * d_c;

	// Allocated device memory
	hipMalloc(&d_a, bytes);
	hipMalloc(&d_b, bytes);
	hipMalloc(&d_c, bytes);

	// Initialize matrices
	matrix_init(h_a, n);
	matrix_init(h_b, n);

	// Copy data to the device
	hipMemcpy(d_a, h_a, bytes, hipMemcpyHostToDevice);
	hipMemcpy(d_b, h_b, bytes, hipMemcpyHostToDevice);

	// Threads per block
	int BLOCK_SIZE = 16;

	// Blocks in each dimension (No padding)
	int GRID_SIZE = n / BLOCK_SIZE;

	// Use dim3 objects
	dim3 grid(GRID_SIZE, GRID_SIZE);
	dim3 threads(BLOCK_SIZE, BLOCK_SIZE);

	// Launch kernel
	matrixMul << <grid, threads >> > (d_a, d_b, d_c, n);

	// Copy back to the host
	hipMemcpy(h_c, d_c, bytes, hipMemcpyDeviceToHost);

	// Free memory on host
	free(h_a);
	free(h_b);
	free(h_c);

	// Free memory on device
	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);

	printf("\nCalculated the matrix successfully...\n");

	// Getting the end time
	auto end = std::chrono::system_clock::now();

	// Calculating elapsed_seconds
	std::chrono::duration<double> elapsed_seconds = end - start;

	// Print elasped time in seconds
	std::cout << "elapsed time: " << elapsed_seconds.count() << "s" << std::endl;

	return 0;
}